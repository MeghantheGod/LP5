#include <hip/hip_runtime.h>

#include <iostream>

__global__ void matmul(int* A, int* B, int* C, int N) {
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    if (Row < N && Col < N) {
        int Pvalue = 0;
        for (int k = 0; k < N; k++) {
            Pvalue += A[Row * N + k] * B[k * N + Col];
        }
        C[Row * N + Col] = Pvalue;
    }
}

int main() {
    int N = 512;
    int size = N * N * sizeof(int);
    int *A = nullptr, *B = nullptr, *C = nullptr;
    int *dev_A = nullptr, *dev_B = nullptr, *dev_C = nullptr;

    // Allocate pinned memory on host
    if (hipHostMalloc((void**)&A, size, hipHostMallocDefault) != hipSuccess ||
        hipHostMalloc((void**)&B, size, hipHostMallocDefault) != hipSuccess ||
        hipHostMalloc((void**)&C, size, hipHostMallocDefault) != hipSuccess) 
    {
        std::cerr << "Failed to allocate pinned host memory!" << std::endl;
        return -1;
    }

    // Allocate memory on device
    if (hipMalloc((void**)&dev_A, size) != hipSuccess ||
        hipMalloc((void**)&dev_B, size) != hipSuccess ||
        hipMalloc((void**)&dev_C, size) != hipSuccess) 
    {
        std::cerr << "Failed to allocate device memory!" << std::endl;
        return -1;
    }

    // Initialize matrices A and B
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            A[i * N + j] = i * N + j;
            B[i * N + j] = j * N + i;
        }
    }

    // Copy matrices to device
    hipMemcpy(dev_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_B, B, size, hipMemcpyHostToDevice);

    // Define block and grid size
    dim3 dimBlock(16, 16);
    dim3 dimGrid((N + dimBlock.x - 1) / dimBlock.x, (N + dimBlock.y - 1) / dimBlock.y);

    // Launch kernel
    matmul<<<dimGrid, dimBlock>>>(dev_A, dev_B, dev_C, N);

    // Synchronize to wait for kernel completion
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(C, dev_C, size, hipMemcpyDeviceToHost);

    // Print a portion of the result matrix
    for (int i = 0; i < 10; i++) {
        for (int j = 0; j < 10; j++) {
            std::cout << C[i * N + j] << " ";
        }
        std::cout << std::endl;
    }

    // Free memory
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);
    hipHostFree(A);
    hipHostFree(B);
    hipHostFree(C);

    return 0;
}
